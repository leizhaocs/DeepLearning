#include "hip/hip_runtime.h"
/**********************************************************************
 *
 * Copyright Lei Zhao.
 * contact: leizhao0403@gmail.com
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 **********************************************************************/

#include "includes.h"

__global__ void dropout_kernel(float *input, float *output, int N, float *rand, float rate)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id < N)
    {
        output[id] = (rand[id] < rate) ? 0 : input[id];
    }
}

/* dropout */
void dropout_gpu(float *input, float *output, int n, int batch, float *rand, float rate)
{
    int N = n * batch;

    dropout_kernel<<<cuda_gridsize(N), BLOCK>>>(input, output, N, rand, rate);
    check_cuda_error();
}

__global__ void backward_dropout_kernel(float *backward_input, float *backward_output, float *rand, float rate, int N)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id < N)
    {
        backward_output[id] = (rand[id] < rate) ? 0 : backward_input[id];
    }
}

/* backward of dropout */
void backward_dropout_gpu(float *backward_input, float *backward_output, float *rand, float rate, int n, int batch)
{
    int N = n * batch;

    backward_dropout_kernel<<<cuda_gridsize(N), BLOCK>>>(backward_input, backward_output, rand, rate, N);
    check_cuda_error();
}

__global__ void maxpool_kernel(int N, int in_h, int in_w, int in_c, int out_h, int out_w,
    int stride_h, int stride_w, int filter_h, int filter_w, int padding_h, int padding_w,
    float *input, float *output, int *index)
{
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= N)
    {
        return;
    }

    int w = id % out_w;
    id /= out_w;
    int h = id % out_h;
    id /= out_h;
    int c = id % in_c;
    id /= in_c;
    int n = id;

    float max = -INFINITY;
    int max_i = -1;
    for (int i = 0; i < filter_h; i++)
    {
        for (int j = 0; j < filter_w; j++)
        {
            int cur_h = h * stride_h + i - padding_h;
            int cur_w = w * stride_w + j - padding_w;

            int in_index = ((n*in_c + c)*in_h + cur_h)*in_w + cur_w;
            int valid = (cur_h >= 0 && cur_h < in_h && cur_w >= 0 && cur_w < in_w);
            float val = (valid != 0) ? input[in_index] : -INFINITY;
            max_i = (val > max) ? in_index : max_i;
            max = (val > max) ? val : max;
        }
    }

    int out_index = ((n*in_c + c)*out_h+ h)*out_w + w;
    output[out_index] = max;
    index[out_index] = max_i;
}

/* max pooling */
void maxpool_gpu(float *input, float *output, int *index, int in_h, int in_w, int in_c, int out_h, int out_w,
    int stride_h, int stride_w, int filter_h, int filter_w, int padding_h, int padding_w, int batch)
{
    int N = out_h * out_w * in_c * batch;

    maxpool_kernel<<<cuda_gridsize(N), BLOCK>>>(N, in_h, in_w, in_c, out_h, out_w,
        stride_h, stride_w, filter_h, filter_w, padding_h, padding_w,
        input, output, index);
    check_cuda_error();
}

__global__ void backward_maxpool_kernel(int N, int in_h, int in_w, int in_c, int out_h, int out_w,
    int stride_h, int stride_w, int filter_h, int filter_w, int padding_h, int padding_w,
    float *backward_input, float *backward_output, int *index)
{
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= N)
    {
        return;
    }

    int in_index = id;
    int w = id % in_w;
    id /= in_w;
    int h = id % in_h;
    id /= in_h;
    int c = id % in_c;
    id /= in_c;
    int n = id;

    float d = 0;
    int area_h = (filter_h - 1) / stride_h;
    int area_w = (filter_w - 1) / stride_w;
    for (int l = -area_h; l < area_h + 1; l++)
    {
        for (int m = -area_w; m < area_w + 1; m++)
        {
            int cur_h = (h + padding_h) / stride_h + l;
            int cur_w = (w + padding_w) / stride_w + m;

            int out_index = ((n*in_c + c)*out_h + cur_h)*out_w + cur_w;
            int valid = (cur_w >= 0 && cur_w < out_w && cur_h >= 0 && cur_h < out_h);
            if (valid && index[out_index] == in_index)
            {
                d += backward_input[out_index];
            }
        }
    }
    backward_output[in_index] += d;
}

/* backward of maxpool */
void backward_maxpool_gpu(float *backward_input, float *backward_output, int *index, int in_h, int in_w, int in_c, int out_h, int out_w,
    int stride_h, int stride_w, int filter_h, int filter_w, int padding_h, int padding_w, int batch)
{
    int N = in_c * in_h * in_w * batch;

    backward_maxpool_kernel<<<cuda_gridsize(N), BLOCK>>>(N, in_h, in_w, in_c, out_h, out_w,
        stride_h, stride_w, filter_h, filter_w, padding_h, padding_w,
        backward_input, backward_output, index);
    check_cuda_error();
}

__global__ void relu_kernel(float *input, float *output, int n)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n)
    {
        output[i] = (input[i] > 0) ? input[i] : 0;
    }
}

/* relu */
void relu_gpu(float *input, float *output, int n)
{
    relu_kernel<<<cuda_gridsize(n), BLOCK>>>(input, output, n);
    check_cuda_error();
}

__global__ void backward_relu_kernel(float *backward_input, float *forward_output, float *backward_output, int n)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n)
    {
        backward_output[i] = (forward_output[i] > 0) ? backward_input[i] : 0;
    }
}

/* backward of relu */
void backward_relu_gpu(float *backward_input, float *forward_output, float *backward_output, int n)
{
    backward_relu_kernel<<<cuda_gridsize(n), BLOCK>>>(backward_input, forward_output, backward_output, n);
    check_cuda_error();
}

__global__ void sigmoid_kernel(float *input, float *output, int n)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n)
    {
        float temp = expf((float)input[i]);
        output[i] = temp / (temp + 1);
    }
}

/* sigmoid */
void sigmoid_gpu(float *input, float *output, int n)
{
    sigmoid_kernel<<<cuda_gridsize(n), BLOCK>>>(input, output, n);
    check_cuda_error();
}

__global__ void backward_sigmoid_kernel(float *backward_input, float *forward_output, float *backward_output, int n)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n)
    {
        backward_output[i] = backward_input[i] * forward_output[i] * (1 - forward_output[i]);
    }
}

/* backward of sigmoid */
void backward_sigmoid_gpu(float *backward_input, float *forward_output, float *backward_output, int n)
{
    backward_sigmoid_kernel<<<cuda_gridsize(n), BLOCK>>>(backward_input, forward_output, backward_output, n);
    check_cuda_error();
}

__global__ void softmax_kernel(float *input, float *output, int n, int batch)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= batch)
    {
        return;
    }

    float *in = input + id*n;
    float *out = output + id*n;

    float largest = -INFINITY;
    for (int i = 0; i < n; i++)
    {
        float val = in[i];
        largest = (val>largest) ? val : largest;
    }
    float sum = 0;
    for (int i = 0; i < n; i++)
    {
        float e = expf(in[i] - largest);
        sum += e;
        out[i] = e;
    }
    for (int i = 0; i < n; i++)
    {
        out[i] /= sum;
    }
}

/* softmax */
void softmax_gpu(float *input, float *output, int n, int batch)
{
    softmax_kernel<<<cuda_gridsize(batch), BLOCK>>>(input, output, n, batch);
    check_cuda_error();
}

__global__ void backward_softmax_kernel(float *backward_input, float *forward_output, float *backward_output, int n, int batch)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= batch)
    {
        return;
    }

    float *backward_in = backward_input + id*n;
    float *forward_out = forward_output + id*n;
    float *backward_out = backward_output + id*n;

    for (int i = 0; i < n; i++)
    {
        backward_out[i] = forward_out[i] - (backward_in[i] != 0);
    }
}

/* backward of softmax */
void backward_softmax_gpu(float *backward_input, float *forward_output, float *backward_output, int n, int batch)
{
    backward_softmax_kernel<<<cuda_gridsize(batch), BLOCK>>>(backward_input, forward_output, backward_output, n, batch);
    check_cuda_error();
}

/* grid size */
dim3 cuda_gridsize(int n)
{
    int k = (n-1) / BLOCK + 1;
    int x = k;
    int y = 1;
    if (x > 65535)
    {
        x = ceil(sqrt(k));
        y = (n-1)/(x*BLOCK) + 1;
    }
    dim3 d = {(unsigned int)x, (unsigned int)y, 1};
    return d;
}

/* check cuda error */
void check_cuda_error()
{                     
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("Cuda failure %s:%d:\n",__FILE__,__LINE__);
        exit(1);
    }
}

/* check cublas error */
void check_cublas_error(hipblasStatus_t status)
{                     
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        printf("Cuda failure %s:%d\n",__FILE__,__LINE__);
        exit(1);
    }
}

/* get blas handle */
hipblasHandle_t blas_handle()
{
    static int init = 0;
    static hipblasHandle_t handle;
    if (!init)
    {
        hipblasCreate(&handle);
        init = 1;
    }
    return handle;
}
